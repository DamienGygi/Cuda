#include "hip/hip_runtime.h"
#include "RayTracing.h"


#include <iostream>
#include <assert.h>

#include "Device.h"
#include <assert.h>

#include "length_cm.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void raytracingGM(uchar4* ptrTabPixels, uint w, uint h, float t, Sphere *ptrSphere, int nbSphere);
extern __global__ void raytracingCM(uchar4* TAB_CM, uint w, uint h, float t, Sphere *ptrSphere, int nbSphere);
extern __host__ void upLoadGPUCM(Sphere* tabValue);
/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\raytracingcm
 |*	Constructeur	    *|
 \*-------------------------*/

RayTracing::RayTracing(const Grid &grid, uint w, uint h, float dt, int nbSphere) :
	Animable_I<uchar4>(grid, w, h, "Raytracing_Cuda")
    {
    assert(nbSphere==LENGTH_CM);
    // time
    this->t = 0;
    this->dt = dt;
    this->nbSphere = nbSphere;

    SphereCreator sphereCreator = SphereCreator(nbSphere, w, h,100);
    Sphere* ptrTabSphere = sphereCreator.getTabSphere();

    //MemoryManagement
    this->sizeOctetSpheres = nbSphere * sizeof(Sphere);
    Device::malloc(&ptrDevTabSphere, sizeOctetSpheres);
    Device::memclear(ptrDevTabSphere, sizeOctetSpheres);
    Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctetSpheres);

    upLoadGPUCM(ptrTabSphere);
    }

RayTracing::~RayTracing()
    {
    Device::free(ptrDevTabSphere);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("raytracing rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    // TODO lancer le kernel avec <<<dg,db>>>
    //raytracingGM<<<dg,db>>>(ptrDevPixels,w,h,t, ptrDevTabSphere, nbSphere);
    raytracingCM<<<dg,db>>>(ptrDevPixels,w,h,t, ptrDevTabSphere, nbSphere);

    // le kernel est importer ci-dessus (ligne 19)

    Device::lastCudaError("raytracing rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
