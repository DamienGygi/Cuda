#include "hip/hip_runtime.h"
#include "Device.h"
#include <iostream>
#include "MonteCarlo.h"

using std::cout;
using std::endl;

extern __global__ void monteCarlo(hiprandState* tabDevGenerator, int nbFleches, float m,int* ptrDevNx);
extern __global__ void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId);

MonteCarlo::MonteCarlo(int nbFleches,int m,const Grid& grid) :
	nbFleches(nbFleches)
    {
    this->db=grid.db;
    this->dg=grid.dg;
    this->m=m;

    this->sizeOctetPi = sizeof(int);

    Device::malloc(&ptrDevNx, sizeOctetPi);
    Device::memclear(ptrDevNx, sizeOctetPi);

    this->sizeOctetSM = db.x * db.y * db.z * sizeof(int);

    int nbThread =grid.threadCounts();
    size_t sizeOctet= nbThread*sizeof(hiprandState) ;
    Device::malloc(&ptrDevGenerator, sizeOctet);
    Device::memclear(ptrDevGenerator, sizeOctet);

    setup_kernel_rand<<<dg,db>>>(ptrDevGenerator, Device::getDeviceId());

    }

MonteCarlo::~MonteCarlo(void)
    {
    Device::free(ptrDevNx);
    Device::free(ptrDevGenerator);
    }

void MonteCarlo::process()
    {
    //Device::lastCudaError("Slice (before)"); // temp debug
    monteCarlo<<<dg,db, sizeOctetSM>>>(ptrDevGenerator, nbFleches,m,ptrDevNx);// assynchrone
    //Device::lastCudaError("Slice (after)"); // temp debug
    Device::memcpyDToH(&piTest, ptrDevNx, sizeOctetPi);
    pi=(float)piTest/nbFleches*m;
    //hipMemcpy(&pi, ptrDevNx, sizeOctetPi, hipMemcpyDeviceToHost); // barriere synchronisation implicite
    //pi /= nbFleches;
    }

float MonteCarlo::getPi()
    {
    return this->pi;
    }
