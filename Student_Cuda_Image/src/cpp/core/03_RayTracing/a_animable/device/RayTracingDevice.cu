#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"

#include "Sphere.h"
#include "RayTracingMath.h"
#include "length_cm.h"

using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl?me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff?rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/
__constant__ float TAB_CM[LENGTH_CM];
/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__host__ void uploadGPUCM(Sphere* tabValue);

__global__ void raytracingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevSphere, int nbSphere);
__global__ void raytracingCM(uchar4* TAB_CM, uint w, uint h, float t, Sphere* ptrDevSphere, int nbSphere);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
static __device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevSphere, int nbSphere);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ void uploadGPUCM(Sphere* tabValue)
    {
    size_t size = LENGTH_CM * sizeof(Sphere);
    int offset = 0;
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(TAB_CM), tabValue, size, offset, hipMemcpyHostToDevice));
    }

__global__ void raytracingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevSphere, int nbSphere)
    {
    work(ptrDevPixels, w, h, t, ptrDevSphere, nbSphere);

    }
__global__ void raytracingCM(uchar4* TAB_CM, uint w, uint h, float t, Sphere* ptrDevSphere, int nbSphere)
    {
    work(TAB_CM, w, h, t, ptrDevSphere, nbSphere);

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
__device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevSphere, int nbSphere)
    {
    RayTracingMath rayTracing = RayTracingMath(ptrDevSphere, nbSphere);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int s = TID;
    int i;
    int j;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);
	rayTracing.colorIJ(&ptrDevPixels[s], i, j, t);
	s += NB_THREAD;
	}

    }
/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
