#include "hip/hip_runtime.h"
#include "Device.h"
#include <iostream>
#include "MonteCarlo.h"

using std::cout;
using std::endl;

extern __global__ void monteCarlo(hiprandState* tabDevGenerator, int nbFleches, float m,int* ptrDevNx);

MonteCarlo::MonteCarlo(int nbFleches,int m,const Grid& grid) :
	nbFleches(nbFleches)
    {
    this->db=grid.db;
    this->dg=grid.dg;

    this->sizeOctetPi = sizeof(int);

    Device::malloc(&ptrDevNx, sizeOctetPi);
    Device::memclear(ptrDevNx, sizeOctetPi);

    this->sizeOctetSM = db.x * db.y * db.z * sizeof(int);

    int nbThread =grid.threadCounts();
    size_t sizeOctet= nbThread*sizeof(hiprandState) ;
    Device::malloc(&ptrDevGenerator, sizeOctet);
    Device::memclear(ptrDevGenerator, sizeOctet);

    }

MonteCarlo::~MonteCarlo(void)
    {
    hipFree (ptrDevNx);
    hipFree (ptrDevGenerator);
    }

void MonteCarlo::process()
    {
    //Device::lastCudaError("Slice (before)"); // temp debug
    monteCarlo<<<dg,db, sizeOctetSM>>>(ptrDevGenerator, nbFleches,m,ptrDevNx);// assynchrone
    //Device::lastCudaError("Slice (after)"); // temp debug
    hipMemcpy(&pi, ptrDevNx, sizeOctetPi, hipMemcpyDeviceToHost); // barriere synchronisation implicite
    //pi /= nbFleches;
    }

float MonteCarlo::getPi()
    {
    return this->pi;
    }
