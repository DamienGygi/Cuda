#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"

#include <stdio.h>


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/


/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void histogramme(int* ptrTabPixelsGM, uint nbPixels, int* ptrGM);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void reductionIntraThread(int* TAB_SM, int* ptrTabPixelsGM, uint nbPixels);

__device__ void reductionInterBlock(int* TAB_SM, int* ptrGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/


/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void histogramme(int* ptrTabPixelsGM, uint nbPixels, int* ptrGM)
    {
    extern __shared__ int TAB_SM[];
    reductionIntraThread(TAB_SM, ptrTabPixelsGM, nbPixels);
    __syncthreads();
    reductionInterBlock(TAB_SM, ptrGM);
    }


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/


__device__ void reductionIntraThread(int* TAB_SM, int* ptrTabPixelsGM, uint nbPixels)
    {
    const int NB_THREAD=Indice2D::nbThread();
    const int TID=Indice2D::tid();
    const int TIDLocal = Indice1D::tidLocal();

    int s = TID;

    while(s<nbPixels)
	{
	int value = ptrTabPixelsGM[s];
	int* adresse = &TAB_SM[value];
	atomicAdd(adresse, 1);
	s += NB_THREAD;
	}
    }

__device__ void reductionInterBlock(int* TAB_SM, int* ptrGM)
    {
	if (Indice2D::tidLocal() == 0)
	    {
	    for (int i = 0; i < 256; i++)
		{
		    ptrGM[i] = TAB_SM[i];
		}
	    }

    }
/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
