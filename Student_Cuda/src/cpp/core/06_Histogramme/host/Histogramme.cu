#include "hip/hip_runtime.h"
#include "../../06_Histogramme/host/Histogramme.h"

#include <iostream>

#include "Device.h"
#include "AleaTools.h"

using std::cout;
using std::endl;


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void histogramme(int* ptrTabPixelsGM, uint nbPixels, int* ptrGM);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Histogramme::Histogramme(const Grid& grid, int* ptrResult, int max)
    {

    this->max = max;
    this->n = max * (max + 1) / 2;
    this->nbShuffles = n * 10;
    this->datas = new int[n];

    dataCreate();
    dataShuffle();

    this->nbPixels= this->n;
    this->data = this->datas;

    this->ptrResult = ptrResult;

    this->sizeOctetResult = sizeof(int) * max;
    this->sizeOctetData = sizeof(int) * this->nbPixels;

    Device::malloc(&ptrTabIn, sizeOctetData);
    Device::memclear(ptrTabIn, sizeOctetData);
    Device::memcpyHToD(ptrTabIn, data, sizeOctetData);
    Device::malloc(&ptrTabOut, sizeOctetResult);
    Device::memclear(ptrTabOut, sizeOctetResult);

    this->dg = grid.dg;
    this->db = grid.db;

    }

Histogramme::~Histogramme(void)
    {
	Device::free(ptrTabOut);
	Device::free(ptrTabIn);
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Histogramme::run()
    {

    histogramme<<<dg,db, sizeOctetResult>>>(ptrTabIn, nbPixels, ptrTabOut);
    Device::synchronize();
    Device::memcpyDToH(ptrResult, ptrTabOut, sizeOctetResult);

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
void Histogramme::dataCreate(void)
    {
    int s = 0;
    for (int i = 0; i < max; i++)
	{
	for (int j = 1; j <= i + 1; j++)
	    {
	    datas[s] = i;
	    //assert(i >= 0 && i <= max);
	    //assert(s < n);
	    s++;
	    }
	}
    //assert(s == n);
    }
void Histogramme::dataShuffle()
    {
    AleaTools aleaTools = AleaTools();
    for (int i = 1; i <= nbShuffles; i++)
	{
	int first = aleaTools.uniformeAB(0, n - 1);
	int second = aleaTools.uniformeAB(0, n - 1);
	swap(first, second);
	}
    }
void Histogramme::swap(int i, int j)
    {
    //assert(i >= 0 && i < n);
    //assert(j >= 0 && j < n);
    int temp = datas[i];
    datas[i] = datas[j];
    datas[j] = temp;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
