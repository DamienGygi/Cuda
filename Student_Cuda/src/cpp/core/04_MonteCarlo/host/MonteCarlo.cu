#include "hip/hip_runtime.h"
#include "Device.h"
#include <iostream>
#include "MonteCarlo.h"

using std::cout;
using std::endl;

extern __global__ void monteCarlo(hiprandState* tabDevGenerator, int nbFleches, float m,int* ptrDevNx);

MonteCarlo::MonteCarlo(int nbFleches,int m,dim3 dg,dim3 db) :
	nbFleches(nbFleches)
    {
    this->db=db;
    this->dg=dg;
    Grid grid(dg,db);
    this->sizeOctetPi = sizeof(int);
    hipMalloc(&ptrDevNx, sizeOctetPi);
    hipMemset(ptrDevNx, 0, sizeOctetPi);
    this->sizeOctetSM = db.x * db.y * db.z * sizeof(int);

    int nbThread =grid.threadCounts();
    size_t sizeOctet= nbThread*sizeof(hiprandState) ;
    hiprandState* ptrDevGenerator=NULL;
    hipMalloc(&ptrDevGenerator, sizeOctet);
    }

MonteCarlo::~MonteCarlo(void)
    {
    hipFree (ptrDevNx);
    }

void MonteCarlo::process()
    {
    //Device::lastCudaError("Slice (before)"); // temp debug
    monteCarlo<<<dg,db, sizeOctetSM>>>(tabDevGenerator, nbFleches,m,ptrDevNx);// assynchrone
    //Device::lastCudaError("Slice (after)"); // temp debug
    hipMemcpy(&pi, ptrDevNx, sizeOctetPi, hipMemcpyDeviceToHost); // barriere synchronisation implicite
    //pi /= nbFleches;
    }

float MonteCarlo::getPi()
    {
    return this->pi;
    }
