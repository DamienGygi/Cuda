#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"
#include "Sphere.h"
#include "RayTracingMath.h"


using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void raytracing(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere, uint nbSphere);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void raytracing(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere, uint nbSphere)
    {
    RayTracingMath rayTracing = RayTracingMath(ptrDevTabSphere, nbSphere);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int s = TID;
    int i;
    int j;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);
	rayTracing.colorIJ(&ptrDevPixels[s], i,j, t);
	s += NB_THREAD;
	}

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

