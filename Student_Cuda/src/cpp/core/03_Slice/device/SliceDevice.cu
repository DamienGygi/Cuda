#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "reductionADD.h"
#include <stdio.h>

__global__ void slice(float *ptrDevPi, int nbSlice);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reduceIntraThread(float* tabSM, int nbSlice);
static __device__ float area(int x, int nbSlice);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void slice(float* ptrDevPi, int nbSlice)
{
	extern __shared__ float tabSM[];
	reduceIntraThread(tabSM, nbSlice);  /* Bloc Orange */
	__syncthreads(); /* Synchronisation des Threads*/
	reductionADD<float>(tabSM,ptrDevPi); /* Bloc bleu + bloc vert */
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__
void reduceIntraThread(float *tabSM, int nbSlice)
{
	/* Shared Memory */
	float sumThread = 0.0f;

	//const int TID_LOCAL = threadIdx.x;
	//const int TID = threadIdx.x+(blockIdx.x*blockDim.x);
	//const int NB_THREAD = blockDim.x*gridDim.x;
	const int TID_LOCAL = Indice1D::tidLocal();
	const int TID = Indice1D::tid();
	const int NB_THREAD = Indice1D::nbThread();
	int s = TID;
	while (s < nbSlice)
	{
		sumThread += area(s, nbSlice);
		s += NB_THREAD;
	}
	tabSM[TID_LOCAL] = sumThread;
	//tabSM[TID_LOCAL] = 1;
	//tabSM[TID_LOCAL] = TID;

}

__device__ float f(float x)
{
	return 4.0f / (1.0f + x * x);
}

__device__ float area(int s, int nbSlice)
{
	return f(s / (float) nbSlice);
	//return (float)s;
}
