#include "hip/hip_runtime.h"
#include "RayTracing.h"


#include <iostream>
#include <assert.h>

#include "Device.h"
#include <assert.h>


using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void raytracing(uchar4* ptrTabPixels, uint w, uint h, float t, Sphere *ptrSphere, int nbSphere);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

RayTracing::RayTracing(const Grid &grid, uint w, uint h, float dt, int nbSphere) :
	Animable_I<uchar4>(grid, w, h, "Raytracing_Cuda")
    {

    // time
    this->t = 0;
    this->dt = dt;
    this->nbSphere = nbSphere;

    SphereCreator sphereCreator = SphereCreator(nbSphere, w, h,100);
    Sphere* ptrTabSphere = sphereCreator.getTabSphere();

    //MemoryManagement
    this->sizeOctetSpheres = nbSphere * sizeof(Sphere);
    Device::malloc(&ptrDevTabSphere, sizeOctetSpheres);
    Device::memclear(ptrDevTabSphere, sizeOctetSpheres);
    Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctetSpheres);

    }

RayTracing::~RayTracing()
    {
    Device::free(ptrDevTabSphere);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("raytracing rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    // TODO lancer le kernel avec <<<dg,db>>>
    raytracing<<<dg,db>>>(ptrDevPixels,w,h,t, ptrDevTabSphere, nbSphere);
    // le kernel est importer ci-dessus (ligne 19)

    Device::lastCudaError("raytracing rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
    {
    //t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
