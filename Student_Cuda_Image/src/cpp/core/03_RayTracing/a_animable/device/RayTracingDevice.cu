#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"

#include "Sphere.h"
#include "RayTracingMath.h"
#include "length_cm.h"

using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl?me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff?rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/
__constant__ Sphere TAB_CM[LENGTH_CM];
/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__host__ void uploadGPUCM(Sphere* tabValue);

__global__ void raytracingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevSphere, int nbSphere);
__global__ void raytracingCM(uchar4* ptrDevPixel, uint w, uint h, float t, Sphere* ptrDevSphere, int nbSphere);
__global__ void raytracingSM(uchar4* ptrDevPixel, uint w, uint h, float t, Sphere* ptrDevSphere, int nbSphere);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
static __device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevSphere, int nbSphere);
static __device__ void copyToSM(Sphere* ptrDevSphere, Sphere* tab_SM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ void uploadGPUCM(Sphere* tabValue)
    {
    size_t size = LENGTH_CM * sizeof(Sphere);
    int offset = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(TAB_CM), tabValue, size, offset, hipMemcpyHostToDevice);
    }

__global__ void raytracingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevSphere, int nbSphere)
    {
    work(ptrDevPixels, w, h, t, ptrDevSphere, nbSphere);

    }
__global__ void raytracingCM(uchar4* ptrDevPixel, uint w, uint h, float t, Sphere* ptrDevSphere, int nbSphere)
    {
    work(ptrDevPixel, w, h, t, ptrDevSphere, nbSphere);
    }
__global__ void raytracingSM(uchar4* ptrDevPixel, uint w, uint h, float t, Sphere* ptrDevSphere, int nbSphere)
    {
    extern __shared__ Sphere tab_SM[];
    copyToSM(ptrDevSphere,tab_SM);

    __syncthreads();
    work(ptrDevPixel, w, h, t, tab_SM, nbSphere);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
__device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevSphere, int nbSphere)
    {
    RayTracingMath rayTracing = RayTracingMath(ptrDevSphere, nbSphere);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int s = TID;
    int i;
    int j;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);
	rayTracing.colorIJ(&ptrDevPixels[s], i, j, t);
	s += NB_THREAD;
	}

    }

__device__ void copyToSM(Sphere* ptrDevSphere,Sphere* tab_SM)
    {
	const int TID_LOCAL = Indice2D::tidLocal();
    	const int NB_THREAD_LOCAL = Indice2D::nbThreadLocal();
    	int s = TID_LOCAL;
    	while (s<LENGTH_CM)
    	{
    		tab_SM[s]=ptrDevSphere[s];
    		s += NB_THREAD_LOCAL;
    	}
    }
/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
