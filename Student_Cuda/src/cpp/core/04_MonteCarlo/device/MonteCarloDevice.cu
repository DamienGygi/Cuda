#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include <Indice1D.h>
#include "reductionADD.h"

__global__ void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId);
__global__ void monteCarlo(hiprandState* tabDevGenerator, int nbFleches, float m,int* ptrDevNx);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
static __device__ float f(float x);
static __device__ void reduceIntraThread(hiprandState* tabDevGenerator,int* tabSM, int nbFleches,int m);
/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__
void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId)
    {
// Customisation du generator:
// Proposition, au lecteur de faire mieux !
// Contrainte : Doit etre différent d'un GPU à l'autre
// Contrainte : Doit etre différent dun thread à lautre
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &tabDevGenerator[TID]);
    }

__global__
void monteCarlo(hiprandState* tabDevGenerator, int nbFleches, float m,int* ptrDevNx)
    {
    extern __shared__ int tabSM[];
    reduceIntraThread(tabDevGenerator, tabSM, nbFleches,m);
    __syncthreads();
    reductionADD<int>(tabSM,ptrDevNx);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
__device__ void reduceIntraThread(hiprandState* tabDevGenerator,int* tabSM, int nbFleches,int m){
    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocal();
    const int NB_THREAD = Indice1D::nbThread();
// Global Memory -> Register (optimization)
    hiprandState localGenerator = tabDevGenerator[TID];
    float xAlea;
    float yAlea;
    float y;
    int nx = 0;
    for (int i = 1; i <= nbFleches / NB_THREAD; i++)
	{
	xAlea = hiprand_uniform(&localGenerator); //Genere des nombres entre 0 et 1
	yAlea = hiprand_uniform(&localGenerator) * m;

	y = f(xAlea);
	if (y >= yAlea)
	    {
	    nx++;
	    }
	//work(xAlea, yAlea);
	}
//Register -> Global Memory
//Necessaire si on veut utiliser notre generator
// - dans dautre kernel
// - avec dautres nombres aleatoires !
    tabDevGenerator[TID] = localGenerator;
    tabSM[TID_LOCAL]=nx;
}

__device__ float f(float x)
    {
    return 4.0f / (1.0f + x * x);
    }

