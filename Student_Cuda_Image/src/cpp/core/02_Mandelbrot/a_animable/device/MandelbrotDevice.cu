#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "MandelbrotMath.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"

using namespace gpu;
using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h,float t,uint n, DomaineMath domaineMath);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * Override (code entrainement cuda)
 */
__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h,float t,uint n, DomaineMath domaineMath)
    {
    MandelbrotMath mandelbrotMath(n); // ici pour preparer cuda

    const int WH = w * h;

    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();

    int i;
    int j;
    double x;
    double y;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[
	domaineMath.toXY(i, j, &x, &y);
	mandelbrotMath.colorXY(&ptrDevPixels[s], x, y, t);
	s += NB_THREAD;
	}

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

