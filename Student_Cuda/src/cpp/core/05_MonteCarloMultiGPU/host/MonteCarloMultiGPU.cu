#include "hip/hip_runtime.h"
#include "MonteCarloMultiGPU.h"
#include "MonteCarlo.h"

#include "Device.h"
#include <iostream>

using std::cout;
using std::endl;

extern __global__ void monteCarlo(hiprandState* tabDevGenerator, int nbFleches, float m,int* ptrDevNx);
extern __global__ void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId);

MonteCarloMultiGPU::MonteCarloMultiGPU(int nbFleches,int m,const Grid& grid) :
	nbFleches(nbFleches)
    {
    this->db=grid.db;
    this->dg=grid.dg;
    this->m=m;
    this->grid=grid;

    this->sizeOctetPi = sizeof(int);

    Device::malloc(&ptrDevNx, sizeOctetPi);
    Device::memclear(ptrDevNx, sizeOctetPi);

    this->sizeOctetSM = db.x * db.y * db.z * sizeof(int);

    int nbThread =grid.threadCounts();
    size_t sizeOctet= nbThread*sizeof(hiprandState) ;
    Device::malloc(&ptrDevGenerator, sizeOctet);
    Device::memclear(ptrDevGenerator, sizeOctet);

    setup_kernel_rand<<<dg,db>>>(ptrDevGenerator, Device::getDeviceId());

    }

MonteCarloMultiGPU::~MonteCarloMultiGPU(void)
    {
    Device::free(ptrDevNx);
    Device::free(ptrDevGenerator);
    }

void MonteCarloMultiGPU::process()
    {
    int nbDevice=Device::getDeviceCount();
    int nbFlechesGPU=nbFleches/nbDevice;
    int totalResult;

    #pragma omp parallel for reduction(+:totalResult)
	for(int deviceId=0;deviceId<nbDevice;deviceId++)
	    {
		Device::setDevice(deviceId);
		MonteCarlo monteCarlo(nbFlechesGPU,m,grid);
		monteCarlo.process();
		totalResult+=monteCarlo.getPi();
	    }
    pi=totalResult;
    //pi /= nbFleches;
    }

float MonteCarloMultiGPU::getPi()
    {
    return this->pi;
    }
