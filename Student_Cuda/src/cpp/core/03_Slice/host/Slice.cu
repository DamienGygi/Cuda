#include "hip/hip_runtime.h"
#include "Slice.h"

#include "Device.h"
#include <iostream>

using std::cout;
using std::endl;

extern __global__ void slice(float* ptrDevPi, int nbSlice);

Slice::Slice(int nbSlice,dim3 dg,dim3 db) :
	nbSlice(nbSlice)
    {
    this->db=db;
    this->dg=dg;
    this->sizeOctetPi = sizeof(float);
    hipMalloc(&ptrDevPi, sizeOctetPi);
    hipMemset(ptrDevPi, 0, sizeOctetPi);
    Device::gridHeuristic(dg, db);
    this->sizeOctetSM = db.x * db.y * db.z * sizeof(float);
    }

Slice::~Slice(void)
    {
    hipFree (ptrDevPi);
    }

void Slice::process()
    {
    //Device::lastCudaError("Slice (before)"); // temp debug
    slice<<<dg,db, sizeOctetSM>>>(ptrDevPi, nbSlice);// assynchrone
    //Device::lastCudaError("Slice (after)"); // temp debug
    hipMemcpy(&pi, ptrDevPi, sizeOctetPi, hipMemcpyDeviceToHost); // barriere synchronisation implicite
    pi /= nbSlice;
    }

float Slice::getPi()
    {
    return this->pi;
    }
