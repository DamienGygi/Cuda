#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Device.h"

#include "Mandelbrot.h"
#include "MandelbrotMath.h"
#include <assert.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/
extern __global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, uint n, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h, float t, uint n, const DomaineMath &domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Mandelbrot_Cuda", domaineMath), variateurAnimation(Interval<float>(30,100), t)
    {
    // Input
    this->n = n;

    // Tools
    this->t = t;					// protected dans super classe Animable

    }

Mandelbrot::~Mandelbrot(void)
    {
    // rien
    }

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * Override
 */
void Mandelbrot::animationStep()
    {
    this->t+=t;
    //this->n =variateurAnimation.varierAndGet();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * Override (code naturel omp)
 */
void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("rippling rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release
    mandelbrot<<<dg,db>>>(ptrDevPixels,w,h,t,n,domaineMath);
    // le kernel est importer ci-dessus (ligne 19)
    Device::lastCudaError("rippling rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

